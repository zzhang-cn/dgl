#include "hip/hip_runtime.h"
/*!
 *  Copyright (c) 2021 by Contributors
 * \file nccl_api.cc
 * \brief Implementation of wrapper around NCCL routines.
 */


#include "nccl_api.h"

#include <dgl/array.h>
#include <dgl/aten/array_ops.h>
#include <dgl/runtime/container.h>
#include <dgl/runtime/device_api.h>
#include <dgl/packed_func_ext.h>
#include <dgl/runtime/registry.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

#include <cmath>
#include <sstream>
#include <iomanip>
#include <utility>
#include <vector>
#include <memory>
#include <string>
#include <limits>

#include "cuda_common.h"
#include "../../kernel/cuda/atomic.cuh"
#include "../../array/cuda/dgl_cub.cuh"
#include "../../array/cuda/array_index_select.cuh"

#define NCCL_CALL(func) \
{ \
  ncclResult_t result = func; \
  if (result != ncclSuccess) { \
      LOG(FATAL)                                                        \
          << "NCCLError: " #func " failed with error: " << result;            \
  } \
}

namespace dgl {

using namespace kernel::cuda;

namespace runtime {
namespace cuda {

namespace {

enum class AllToAllMode : int {
  REMAINDER = 0
};


template<typename T> ncclDataType_t NCCLType();
template<> ncclDataType_t NCCLType<int32_t>() {
    return ncclInt32;
}
template<> ncclDataType_t NCCLType<int64_t>() {
    return ncclInt64;
}
template<> ncclDataType_t NCCLType<__half>() {
    return ncclHalf;
}
template<> ncclDataType_t NCCLType<float>() {
    return ncclFloat32;
}
template<> ncclDataType_t NCCLType<double>() {
    return ncclFloat64;
}


template<typename IdType> __global__ void _MapProcByRemainder(
    const IdType * const index,
    const int64_t num_index,
    const int64_t num_proc,
    IdType * const proc_id) {
  const int64_t idx = blockDim.x*static_cast<int64_t>(blockIdx.x)+threadIdx.x;

  if (idx < num_index) {
    proc_id[idx] = index[idx] % num_proc;
  }
}

template<typename IdType>
__global__ void _MapProcByMaskRemainder(
    const IdType * const index,
    const int64_t num_index,
    const IdType mask,
    IdType * const proc_id) {
  const int64_t idx = blockDim.x*static_cast<int64_t>(blockIdx.x)+threadIdx.x;

  if (idx < num_index) {
    proc_id[idx] = index[idx] & mask;
  }
}

template<typename IdType, typename DType>
__global__ void _DualPermKernel(
    const IdType * const in_idx,
    const DType * const in_value,
    const IdType * const perm,
    const int64_t num_in,
    const int64_t num_feat,
    IdType * const out_idx,
    DType * const out_value) {
  // set index permutation
  const int64_t tidx = blockDim.x*static_cast<int64_t>(blockIdx.x)+threadIdx.x;
  if (tidx < num_in) {
    const IdType perm_idx = perm[tidx];
    assert(perm_idx < num_in);
    out_idx[tidx] = in_idx[perm_idx];
  }

  if (num_feat > 1) {
    for (int d = 0; d < blockDim.x; ++d) {
      const int64_t bidx = blockDim.x*static_cast<int64_t>(blockIdx.x) + d;
      if (bidx < num_in) {
        const IdType perm_idx = perm[bidx];
        for (int64_t f = threadIdx.x; f < num_feat; f+=blockDim.x) {
          out_value[bidx*num_feat+f] = in_value[perm_idx*num_feat+f];
        }
      }
    }
  } else {
    if (tidx < num_in) {
      const IdType perm_idx = perm[tidx];
      out_value[tidx] = in_value[perm_idx];
    }
  }
}

template<typename IdType>
__global__ void _ConvertToLocalByRemainder(
    IdType * const items,
    const int64_t num_items,
    const int comm_size) {
  const int64_t idx = threadIdx.x+blockDim.x*blockIdx.x;

  if (idx < num_items) {
    items[idx] = items[idx] / comm_size;
  }
}

template <typename DType, typename IdType>
__global__ void _InversePermKernel(
        const DType* const array,
        const int64_t num_feat,
        int64_t length,
        const IdType* const perm,
        DType* const out) {
  int64_t in_row = blockIdx.x*blockDim.y+threadIdx.y;

  const int64_t stride = blockDim.y*gridDim.x;

  while (in_row < length) {
    int64_t col = threadIdx.x;
    const int64_t out_row = perm[in_row];
    while (col < num_feat) {
      out[out_row*num_feat+col] = array[in_row*num_feat+col];
      col += blockDim.x;
    }
    in_row += stride;
  }
}


}  // namespace

/* NCCLUniqueId **************************************************************/

NCCLUniqueId::NCCLUniqueId() :
  id_() {
  // this ID is unique to the process, not to each call of this function
  NCCL_CALL(ncclGetUniqueId(&id_));
}

ncclUniqueId NCCLUniqueId::Get() const {
  return id_;
}

std::string NCCLUniqueId::ToString() const {
  std::ostringstream oss;

  oss << std::hex;

  for (size_t b = 0; b < NCCL_UNIQUE_ID_BYTES; ++b) {
    const int num = static_cast<uint8_t>(id_.internal[b]);
    oss << std::setw(2) << std::setfill('0') << num;
  }

  std::string result = oss.str();
  CHECK_EQ(result.length(), NCCL_UNIQUE_ID_BYTES*2) <<
    "Invalid NCCL ID format: '" << result << "'";

  return result;
}

void NCCLUniqueId::FromString(
    const std::string& str) {
  // must be exactly 256 hex characters
  CHECK_EQ(str.length(), NCCL_UNIQUE_ID_BYTES * 2) <<
        "Invalid NCCL ID format: '" << str << "'";

  for (size_t b = 0; b < NCCL_UNIQUE_ID_BYTES; ++b) {
    id_.internal[b] = std::strtol(str.substr(b*2, 2).c_str(), nullptr, 16);
  }
}

template<typename IdType>
void GenerateSparseBufferFromRemainder(
    DeviceAPI* const device,
    const DGLContext& ctx,
    const int64_t comm_size,
    const int64_t num_in,
    const IdType * const in_idx,
    IdType * const out_idx,
    IdType * const out_perm,
    int64_t * const out_counts,
    hipStream_t stream) {
  const int64_t comm_bits =
      static_cast<int64_t>(std::ceil(std::log2(comm_size)));

  // this should only run when we have things to send, otherwise comm_bits
  // will be zero, and several operations will fail
  CHECK_GT(comm_size, 1);

  CUDA_CALL(hipMemsetAsync(
      out_counts, 0, sizeof(*out_counts)*(comm_size+1), stream));

  if (num_in == 0) {
    // now that we've zero'd out_counts, nothing left to do
    return;
  }

  // First, generate a mapping of indexes to processors
  IdType * proc_id_in = static_cast<IdType*>(
      device->AllocWorkspace(ctx, sizeof(IdType)*num_in));
  {
    const dim3 block(256);
    const dim3 grid((num_in+block.x-1)/block.x);

    if (comm_size < (1 << comm_bits)) {
      // comm_size is not a power of 2
      _MapProcByRemainder<<<grid, block, 0, stream>>>(
          in_idx,
          num_in,
          comm_size,
          proc_id_in);
      CUDA_CALL(hipGetLastError());
    } else {
      // comm_size is a power of 2
      _MapProcByMaskRemainder<<<grid, block, 0, stream>>>(
          in_idx,
          num_in,
          static_cast<IdType>(comm_size-1),  // bit mask
          proc_id_in);
      CUDA_CALL(hipGetLastError());
    }
  }

  // then create a permutation array that groups processors together by
  // performing a radix sort
  IdType * proc_id_out = static_cast<IdType*>(
      device->AllocWorkspace(ctx, sizeof(IdType)*num_in));
  {
    IdArray perm_in = aten::Range(0, num_in, sizeof(IdType)*8, ctx);

    size_t sort_workspace_size;
    CUDA_CALL(hipcub::DeviceRadixSort::SortPairs(nullptr, sort_workspace_size,
        proc_id_in, proc_id_out, static_cast<IdType*>(perm_in->data), out_perm,
        num_in, 0, comm_bits, stream));

    void * sort_workspace = device->AllocWorkspace(ctx, sort_workspace_size);
    CUDA_CALL(hipcub::DeviceRadixSort::SortPairs(sort_workspace, sort_workspace_size,
        proc_id_in, proc_id_out, static_cast<IdType*>(perm_in->data), out_perm,
        num_in, 0, comm_bits, stream));
    device->FreeWorkspace(ctx, sort_workspace);
  }
  device->FreeWorkspace(ctx, proc_id_in);

  // finally, permute the input arrays
  // sort the data into continuous buffers for sending
  {
    const dim3 block(256);
    const dim3 grid((num_in+block.x-1)/block.x);

    aten::impl::IndexSelectSingleKernel<<<grid, block, 0, stream>>>(
        in_idx,
        out_perm,
        num_in,
        out_idx);
    CUDA_CALL(hipGetLastError());
  }

  // Count the number of values to be sent to each processor
  {
    using AtomicCount = unsigned long long; // NOLINT
    static_assert(sizeof(AtomicCount) == sizeof(int64_t),
        "AtomicCount must be the same width as int64_t for atomicAdd "
        "in hipcub::DeviceHistogram::HistogramEven() to work");

    // TODO(dlasalle): Once https://github.com/NVIDIA/cub/pull/287 is merged,
    // add a compile time check against the cub version to allow
    // num_in > (2 << 31).
    CHECK(num_in < static_cast<int64_t>(std::numeric_limits<int>::max())) <<
        "number of values to insert into histogram must be less than max "
        "value of int.";

    size_t hist_workspace_size;
    CUDA_CALL(hipcub::DeviceHistogram::HistogramEven(
        nullptr,
        hist_workspace_size,
        proc_id_out,
        reinterpret_cast<AtomicCount*>(out_counts),
        comm_size+1,
        static_cast<IdType>(0),
        static_cast<IdType>(comm_size+1),
        static_cast<int>(num_in),
        stream));

    void * hist_workspace = device->AllocWorkspace(ctx, hist_workspace_size);
    CUDA_CALL(hipcub::DeviceHistogram::HistogramEven(
        hist_workspace,
        hist_workspace_size,
        proc_id_out,
        reinterpret_cast<AtomicCount*>(out_counts),
        comm_size+1,
        static_cast<IdType>(0),
        static_cast<IdType>(comm_size+1),
        static_cast<int>(num_in),
        stream));
    device->FreeWorkspace(ctx, hist_workspace);
  }
  device->FreeWorkspace(ctx, proc_id_out);
}

template<typename IdType, typename DType>
void GenerateSparseBuffersFromRemainder(
    DeviceAPI* const device,
    const DGLContext& ctx,
    const int64_t comm_size,
    const int64_t num_in,
    const int64_t num_feat,
    const IdType * const in_idx,
    const DType * const in_value,
    IdType * const out_idx,
    DType * const out_value,
    int64_t * const out_counts,
    hipStream_t stream) {
  const int64_t comm_bits =
      static_cast<int64_t>(std::ceil(std::log2(comm_size)));

  // this should only run when we have things to send, otherwise comm_bits
  // will be zero, and several operations will fail
  CHECK_GT(comm_size, 1);

  CUDA_CALL(hipMemsetAsync(
      out_counts, 0, sizeof(*out_counts)*(comm_size+1), stream));

  if (num_in == 0) {
    // now that we've zero'd out_counts, nothing left to do
    return;
  }

  // First, generate a mapping of indexes to processors
  IdType * proc_id_in = static_cast<IdType*>(
      device->AllocWorkspace(ctx, sizeof(IdType)*num_in));
  {
    const dim3 block(256);
    const dim3 grid((num_in+block.x-1)/block.x);

    if (comm_size < (1 << comm_bits)) {
      // comm_size is not a power of 2
      _MapProcByRemainder<<<grid, block, 0, stream>>>(
          in_idx,
          num_in,
          comm_size,
          proc_id_in);
      CUDA_CALL(hipGetLastError());
    } else {
      // comm_size is a power of 2
      _MapProcByMaskRemainder<<<grid, block, 0, stream>>>(
          in_idx,
          num_in,
          static_cast<IdType>(comm_size-1),  // bit mask
          proc_id_in);
      CUDA_CALL(hipGetLastError());
    }
  }

  // then create a permutation array that groups processors together by
  // performing a radix sort
  IdType * proc_id_out = static_cast<IdType*>(
      device->AllocWorkspace(ctx, sizeof(IdType)*num_in));
  IdType * perm_out = static_cast<IdType*>(device->AllocWorkspace(ctx,
          sizeof(IdType)*num_in));
  {
    IdArray perm_in = aten::Range(0, num_in, sizeof(IdType)*8, ctx);

    size_t sort_workspace_size;
    CUDA_CALL(hipcub::DeviceRadixSort::SortPairs(nullptr, sort_workspace_size,
        proc_id_in, proc_id_out, static_cast<IdType*>(perm_in->data), perm_out,
        num_in, 0, comm_bits, stream));

    void * sort_workspace = device->AllocWorkspace(ctx, sort_workspace_size);
    CUDA_CALL(hipcub::DeviceRadixSort::SortPairs(sort_workspace, sort_workspace_size,
        proc_id_in, proc_id_out, static_cast<IdType*>(perm_in->data), perm_out,
        num_in, 0, comm_bits, stream));
    device->FreeWorkspace(ctx, sort_workspace);
  }
  device->FreeWorkspace(ctx, proc_id_in);

  // perform a histogram and then prefixsum on the sorted proc_id vector

  // finally, permute the input arrays
  // sort the data into continuous buffers for sending
  {
    const dim3 block(256);
    const dim3 grid((num_in+block.x-1)/block.x);

    _DualPermKernel<<<grid, block, 0, stream>>>(
        in_idx,
        in_value,
        perm_out,
        num_in,
        num_feat,
        out_idx,
        out_value);
    CUDA_CALL(hipGetLastError());
  }
  device->FreeWorkspace(ctx, perm_out);

  // Count the number of values to be sent to each processor
  {
    using AtomicCount = unsigned long long; // NOLINT
    static_assert(sizeof(AtomicCount) == sizeof(int64_t),
        "AtomicCount must be the same width as int64_t for atomicAdd "
        "in hipcub::DeviceHistogram::HistogramEven() to work");

    // TODO(dlasalle): Once https://github.com/NVIDIA/cub/pull/287 is merged,
    // add a compile time check against the cub version to allow
    // num_in > (2 << 31).
    CHECK(num_in < static_cast<int64_t>(std::numeric_limits<int>::max())) <<
        "number of values to insert into histogram must be less than max "
        "value of int.";

    size_t hist_workspace_size;
    CUDA_CALL(hipcub::DeviceHistogram::HistogramEven(
        nullptr,
        hist_workspace_size,
        proc_id_out,
        reinterpret_cast<AtomicCount*>(out_counts),
        comm_size+1,
        static_cast<IdType>(0),
        static_cast<IdType>(comm_size+1),
        static_cast<int>(num_in),
        stream));

    void * hist_workspace = device->AllocWorkspace(ctx, hist_workspace_size);
    CUDA_CALL(hipcub::DeviceHistogram::HistogramEven(
        hist_workspace,
        hist_workspace_size,
        proc_id_out,
        reinterpret_cast<AtomicCount*>(out_counts),
        comm_size+1,
        static_cast<IdType>(0),
        static_cast<IdType>(comm_size+1),
        static_cast<int>(num_in),
        stream));
    device->FreeWorkspace(ctx, hist_workspace);
  }
  device->FreeWorkspace(ctx, proc_id_out);
}

template<typename IdType, typename DType>
std::pair<IdArray, NDArray> SparsePush(
    NCCLCommunicatorRef comm,
    IdArray in_idx,
    NDArray in_value,
    const int mode_id) {
  CHECK_EQ(in_idx->shape[0], in_value->shape[0]);

  const auto& ctx = in_idx->ctx;
  CHECK_EQ(ctx, in_value->ctx);
  auto device = DeviceAPI::Get(ctx);

  // TODO(dlasalle): Get the stream from the device context.
  hipStream_t stream = 0;

  CHECK_EQ(in_idx->ndim, 1);

  const int64_t num_in = in_idx->shape[0];
  int64_t num_feat = 1;
  for (int d = 1; d < in_value->ndim; ++d) {
    num_feat *= in_value->shape[d];
  }

  const int64_t comm_size = comm->size();

  if (comm_size == 1) {
    // nothing to do, just return original arrays
    return std::pair<IdArray, NDArray>(in_idx, in_value);
  }

  IdType * send_idx = static_cast<IdType*>(device->AllocWorkspace(ctx,
      num_in*sizeof(IdType)));
  DType * send_value = static_cast<DType*>(device->AllocWorkspace(ctx,
      num_in*num_feat*sizeof(DType)));
  int64_t * send_sum = static_cast<int64_t*>(device->AllocWorkspace(ctx,
      (comm_size+1)*sizeof(int64_t)));

  CHECK_EQ(mode_id, static_cast<int>(AllToAllMode::REMAINDER));
  GenerateSparseBuffersFromRemainder(
      device,
      ctx,
      comm_size,
      num_in,
      num_feat,
      static_cast<const IdType*>(in_idx->data),
      static_cast<const DType*>(in_value->data),
      send_idx,
      send_value,
      send_sum,
      stream);

  // compute the prefix sum of the send values
  int64_t * send_prefix = static_cast<int64_t*>(
      device->AllocWorkspace(ctx, sizeof(int64_t)*(comm_size+1)));
  {
    size_t prefix_workspace_size;
    CUDA_CALL(hipcub::DeviceScan::ExclusiveSum(nullptr, prefix_workspace_size,
        send_sum, send_prefix, comm_size+1, stream));

    void * prefix_workspace = device->AllocWorkspace(
        ctx, prefix_workspace_size);
    CUDA_CALL(hipcub::DeviceScan::ExclusiveSum(prefix_workspace, prefix_workspace_size,
        send_sum, send_prefix, comm_size+1, stream));
    device->FreeWorkspace(ctx, prefix_workspace);
  }

  std::vector<int64_t> send_prefix_host(comm_size+1);
  device->CopyDataFromTo(
      send_prefix,
      0,
      send_prefix_host.data(),
      0,
      send_prefix_host.size()*sizeof(*send_prefix),
      ctx,
      DGLContext{kDLCPU, 0},
      DGLType{kDLInt, sizeof(*send_prefix)*8, 1},
      stream);
  device->FreeWorkspace(ctx, send_prefix);

  CHECK_EQ(send_prefix_host.back(), num_in);

  // communicate the amount to send
  int64_t * recv_sum = static_cast<int64_t*>(
      device->AllocWorkspace(ctx, sizeof(int64_t)*(comm_size+1)));
  comm->AllToAll(send_sum, recv_sum, 1, stream);
  device->FreeWorkspace(ctx, send_sum);

  // compute the prefix sum of the recv values
  int64_t * recv_prefix = static_cast<int64_t*>(
      device->AllocWorkspace(ctx, sizeof(int64_t)*(comm_size+1)));
  {
    size_t prefix_workspace_size;
    CUDA_CALL(hipcub::DeviceScan::ExclusiveSum(nullptr, prefix_workspace_size,
        recv_sum, recv_prefix, comm_size+1));

    void * prefix_workspace = device->AllocWorkspace(
        ctx, prefix_workspace_size);
    CUDA_CALL(hipcub::DeviceScan::ExclusiveSum(prefix_workspace, prefix_workspace_size,
        recv_sum, recv_prefix, comm_size+1));
    device->FreeWorkspace(ctx, prefix_workspace);
  }
  device->FreeWorkspace(ctx, recv_sum);

  // finally copy the prefixsum sum down to the host
  std::vector<int64_t> recv_prefix_host(comm_size+1);
  device->CopyDataFromTo(
      recv_prefix,
      0,
      recv_prefix_host.data(),
      0,
      recv_prefix_host.size()*sizeof(*recv_prefix),
      ctx,
      DGLContext{kDLCPU, 0},
      DGLType{kDLInt, sizeof(*recv_prefix)*8, 1},
      stream);
  device->FreeWorkspace(ctx, recv_prefix);

  // use an event to track when copying is done
  hipEvent_t d2h;
  hipEventCreate(&d2h);
  hipEventRecord(d2h, stream);

  // allocate output space
  hipEventSynchronize(d2h);
  hipEventDestroy(d2h);

  IdArray recv_idx = aten::NewIdArray(recv_prefix_host.back(), ctx, sizeof(IdType)*8);

  std::vector<int64_t> value_shape(in_value->ndim, 0);
  value_shape[0] = recv_prefix_host.back();
  for (int d = 1; d < in_value->ndim; ++d) {
    value_shape[d] = in_value->shape[d];
  }
  NDArray recv_value = NDArray::Empty(value_shape, in_value->dtype, ctx);

  // send data
  comm->SparseAllToAll(
      send_idx,
      send_value,
      num_feat,
      send_prefix_host.data(),
      static_cast<IdType*>(recv_idx->data),
      static_cast<DType*>(recv_value->data),
      recv_prefix_host.data(),
      stream);
  device->FreeWorkspace(ctx, send_idx);
  device->FreeWorkspace(ctx, send_value);

  return std::pair<IdArray, NDArray>(recv_idx, recv_value);
}

template<typename IdType, typename DType>
NDArray SparsePull(
    NCCLCommunicatorRef comm,
    IdArray req_idx,
    NDArray local_tensor,
    const int mode_id) {
  const auto& ctx = req_idx->ctx;
  CHECK_EQ(ctx, local_tensor->ctx);
  auto device = DeviceAPI::Get(ctx);

  // TODO(dlasalle): Get the stream from the device context.
  hipStream_t stream = 0;

  CHECK_EQ(req_idx->ndim, 1);

  const int64_t num_in = req_idx->shape[0];
  int64_t num_feat = 1;
  for (int d = 1; d < local_tensor->ndim; ++d) {
    num_feat *= local_tensor->shape[d];
  }

  const int64_t comm_size = comm->size();

  if (comm_size == 1) {
    // Just return index selection from current local_tensor
    return aten::IndexSelect(local_tensor, req_idx);
  }

  // First we need to send our requests to other processors. This means
  // re-ordering our index array to be contiguous among processors, and
  // counting the number of indices we are sending each processor. For now,
  // we assume a poorly partitioned graph, and that there exists the
  // possibility that each processor could request data from this one.

  // the buffer for us to re-order our requests in
  IdType * send_idx = static_cast<IdType*>(device->AllocWorkspace(ctx,
      num_in*sizeof(IdType)));
  IdType * perm = static_cast<IdType*>(
      device->AllocWorkspace(ctx, sizeof(IdType)*num_in));

  // the number of indexes we need to send to each processor
  int64_t * send_sum = static_cast<int64_t*>(device->AllocWorkspace(ctx,
      (comm_size+1)*sizeof(int64_t)));

  CHECK_EQ(mode_id, static_cast<int>(AllToAllMode::REMAINDER));
  GenerateSparseBufferFromRemainder(
      device,
      ctx,
      comm_size,
      num_in,
      static_cast<const IdType*>(req_idx->data),
      send_idx,
      perm,
      send_sum,
      stream);

  // compute the prefix sum of the indexes this process is requesting
  int64_t * request_prefix = static_cast<int64_t*>(
      device->AllocWorkspace(ctx, sizeof(int64_t)*(comm_size+1)));
  {
    size_t prefix_workspace_size;
    CUDA_CALL(hipcub::DeviceScan::ExclusiveSum(nullptr, prefix_workspace_size,
        send_sum, request_prefix, comm_size+1, stream));

    void * prefix_workspace = device->AllocWorkspace(
        ctx, prefix_workspace_size);
    CUDA_CALL(hipcub::DeviceScan::ExclusiveSum(prefix_workspace, prefix_workspace_size,
        send_sum, request_prefix, comm_size+1, stream));
    device->FreeWorkspace(ctx, prefix_workspace);
  }

  std::vector<int64_t> request_prefix_host(comm_size+1);
  device->CopyDataFromTo(
      request_prefix,
      0,
      request_prefix_host.data(),
      0,
      request_prefix_host.size()*sizeof(*request_prefix),
      ctx,
      DGLContext{kDLCPU, 0},
      DGLType{kDLInt, sizeof(*request_prefix)*8, 1},
      stream);
  device->FreeWorkspace(ctx, request_prefix);
  CHECK_EQ(request_prefix_host.back(), num_in);

  // communicate the amount requested
  int64_t * recv_sum = static_cast<int64_t*>(
      device->AllocWorkspace(ctx, sizeof(int64_t)*(comm_size+1)));
  comm->AllToAll(send_sum, recv_sum, 1, stream);
  device->FreeWorkspace(ctx, send_sum);

  // compute the prefix sum of the requested indexes
  int64_t * response_prefix = static_cast<int64_t*>(
      device->AllocWorkspace(ctx, sizeof(int64_t)*(comm_size+1)));
  {
    size_t prefix_workspace_size;
    CUDA_CALL(hipcub::DeviceScan::ExclusiveSum(nullptr, prefix_workspace_size,
        recv_sum, response_prefix, comm_size+1, stream));

    void * prefix_workspace = device->AllocWorkspace(
        ctx, prefix_workspace_size);
    CUDA_CALL(hipcub::DeviceScan::ExclusiveSum(prefix_workspace, prefix_workspace_size,
        recv_sum, response_prefix, comm_size+1, stream));
    device->FreeWorkspace(ctx, prefix_workspace);
  }
  device->FreeWorkspace(ctx, recv_sum);

  // finally copy the prefixsum sum down to the host
  std::vector<int64_t> response_prefix_host(comm_size+1);
  device->CopyDataFromTo(
      response_prefix,
      0,
      response_prefix_host.data(),
      0,
      response_prefix_host.size()*sizeof(*response_prefix),
      ctx,
      DGLContext{kDLCPU, 0},
      DGLType{kDLInt, sizeof(*response_prefix)*8, 1},
      stream);
  device->FreeWorkspace(ctx, response_prefix);

  // use an event to track when copying is done
  hipEvent_t d2h;
  hipEventCreate(&d2h);
  hipEventRecord(d2h, stream);

  // allocate output space
  hipEventSynchronize(d2h);
  hipEventDestroy(d2h);

  // gather requested indexes
  IdType * recv_idx = static_cast<IdType*>(
      device->AllocWorkspace(ctx, response_prefix_host.back()*sizeof(IdType)));
  comm->AllToAllV(
      send_idx,
      request_prefix_host.data(),
      recv_idx,
      response_prefix_host.data(),
      stream);
  device->FreeWorkspace(ctx, send_idx);

  // convert requested indices to local indices depending on partition
  if (response_prefix_host.back() > 0) {
    const dim3 block(128);
    const dim3 grid((response_prefix_host.back()+block.x-1)/block.x);
    _ConvertToLocalByRemainder<<<grid, block, 0, stream>>>(
        recv_idx, response_prefix_host.back(), comm_size);
  }

  // and then index select them into place
  DType * filled_response_value = static_cast<DType*>(device->AllocWorkspace(ctx,
      response_prefix_host.back()*num_feat*sizeof(DType)));
  if (request_prefix_host.back() > 0) {
    dim3 block(256, 1);
    while (block.x >= 2*num_feat) {
        block.x /= 2;
        block.y *= 2;
    }
    const dim3 grid((request_prefix_host.back()+block.y-1)/block.y);

    aten::impl::IndexSelectMultiKernel<<<grid, block, 0, stream>>>(
        static_cast<const DType*>(local_tensor->data),
        num_feat,
        recv_idx,
        response_prefix_host.back(),
        filled_response_value);
    CUDA_CALL(hipGetLastError());
  }
  device->FreeWorkspace(ctx, recv_idx);

  // we will collect recieved values in this array
  std::vector<int64_t> value_shape(local_tensor->ndim, 0);
  value_shape[0] = request_prefix_host.back();
  for (int d = 1; d < local_tensor->ndim; ++d) {
    value_shape[d] = local_tensor->shape[d];
  }
  DType* filled_request_value = static_cast<DType*>(device->AllocWorkspace(ctx,
      request_prefix_host.back()*num_feat*sizeof(DType)));

  // multiply the prefixes by the number of features being sent
  for (auto& v : request_prefix_host) {
    v *= num_feat;
  }
  for (auto& v : response_prefix_host) {
    v *= num_feat;
  }

  // send the values
  comm->AllToAllV(
      filled_response_value,
      response_prefix_host.data(),
      filled_request_value,
      request_prefix_host.data(),
      stream);
  device->FreeWorkspace(ctx, filled_response_value);

  // finally, we need to permute the values back into the requested order
  NDArray result = NDArray::Empty(value_shape, local_tensor->dtype, ctx);
  if (num_in > 0) {
    dim3 block(256, 1);
    while (block.x >= 2*num_feat) {
        block.x /= 2;
        block.y *= 2;
    }
    const dim3 grid((num_in+block.y-1)/block.y);

    _InversePermKernel<<<grid, block, 0, stream>>>(
        filled_request_value,
        num_feat,
        num_in,
        perm,
        static_cast<DType*>(result->data));
    CUDA_CALL(hipGetLastError());
  }
  device->FreeWorkspace(ctx, filled_request_value);
  device->FreeWorkspace(ctx, perm);

  return result;
}



/* NCCLCommunicator **********************************************************/

NCCLCommunicator::NCCLCommunicator(
    const int size,
    const int rank,
    ncclUniqueId id) :
  comm_(),
  size_(size),
  rank_(rank) {
  CHECK_LT(rank, size);
  CHECK_GE(rank, 0);

  NCCL_CALL(ncclCommInitRank(&comm_, size_, id, rank_));
}

NCCLCommunicator::~NCCLCommunicator() {
  ncclCommDestroy(comm_);
}

ncclComm_t NCCLCommunicator::Get() {
  return comm_;
}

template<typename DType>
void NCCLCommunicator::AllToAllV(
    const DType * const send,
    const int64_t * const send_prefix,
    DType * const recv,
    const int64_t * const recv_prefix,
    hipStream_t stream) {
  const ncclDataType_t type = NCCLType<DType>();

  NCCL_CALL(ncclGroupStart());
  for (int r = 0; r < size_; ++r) {
    const int64_t send_size = send_prefix[r+1]-send_prefix[r];
    if (send_size > 0) {
      NCCL_CALL(ncclSend(send+send_prefix[r], send_size, type, r, comm_, stream));
    }
    const int64_t recv_size = recv_prefix[r+1]-recv_prefix[r];
    if (recv_size > 0) {
      NCCL_CALL(ncclRecv(recv+recv_prefix[r], recv_size, type, r, comm_, stream));
    }
  }
  NCCL_CALL(ncclGroupEnd());
}

template
void NCCLCommunicator::AllToAllV<int32_t>(
    const int32_t * const send,
    const int64_t * send_prefix,
    int32_t * const recv,
    const int64_t * recv_prefix,
    hipStream_t stream);
template
void NCCLCommunicator::AllToAllV<int64_t>(
    const int64_t * const send,
    const int64_t * send_prefix,
    int64_t * const recv,
    const int64_t * recv_prefix,
    hipStream_t stream);
template
void NCCLCommunicator::AllToAllV<float>(
    const float * const send,
    const int64_t * send_prefix,
    float * const recv,
    const int64_t * recv_prefix,
    hipStream_t stream);
template
void NCCLCommunicator::AllToAllV<__half>(
    const __half * const send,
    const int64_t * send_prefix,
    __half * const recv,
    const int64_t * recv_prefix,
    hipStream_t stream);





template<typename IdType>
void NCCLCommunicator::AllToAll(
    const IdType * const send,
    IdType * const recv,
    const int64_t count,
    hipStream_t stream) {
  const ncclDataType_t type = NCCLType<IdType>();

  ncclGroupStart();
  for (int r = 0; r < size_; ++r) {
    ncclSend(send+(r*count), count, type, r, comm_, stream);
    ncclRecv(recv+(r*count), count, type, r, comm_, stream);
  }
  ncclGroupEnd();
}

template
void NCCLCommunicator::AllToAll<int32_t>(
    const int32_t * const send,
    int32_t * const recv,
    const int64_t count,
    hipStream_t stream);
template
void NCCLCommunicator::AllToAll<int64_t>(
    const int64_t * const send,
    int64_t * const recv,
    const int64_t count,
    hipStream_t stream);


template<typename IdType, typename DType>
void NCCLCommunicator::SparseAllToAll(
      const IdType * const send_idx,
      const DType * const send_value,
      const int64_t num_feat,
      const int64_t * const send_prefix,
      IdType * const recv_idx,
      DType * const recv_value,
      const int64_t * const recv_prefix,
      hipStream_t stream) {
  const ncclDataType_t idx_type = NCCLType<IdType>();
  const ncclDataType_t value_type = NCCLType<DType>();

  ncclGroupStart();
  for (int r = 0; r < size_; ++r) {
    const int64_t send_size = send_prefix[r+1]-send_prefix[r];
    if (send_size > 0) {
      ncclSend(send_idx+send_prefix[r], send_size, idx_type, r, comm_, stream);
      ncclSend(send_value+send_prefix[r]*num_feat, send_size*num_feat,
               value_type, r, comm_, stream);
    }
    const int64_t recv_size = recv_prefix[r+1]-recv_prefix[r];
    if (recv_size > 0) {
      ncclRecv(recv_idx+recv_prefix[r], recv_size, idx_type, r, comm_, stream);
      ncclRecv(recv_value+recv_prefix[r]*num_feat, recv_size*num_feat,
               value_type, r, comm_, stream);
    }
  }
  ncclGroupEnd();
}

template
void NCCLCommunicator::SparseAllToAll<int32_t, __half>(
      const int32_t * const send_idx,
      const __half * const send_value,
      const int64_t num_feat,
      const int64_t * const send_prefix,
      int32_t * const recv_idx,
      __half * const recv_value,
      const int64_t * const recv_prefix,
      hipStream_t stream);
template
void NCCLCommunicator::SparseAllToAll<int64_t, __half>(
      const int64_t * const send_idx,
      const __half * const send_value,
      const int64_t num_feat,
      const int64_t * const send_prefix,
      int64_t * const recv_idx,
      __half * const recv_value,
      const int64_t * const recv_prefix,
      hipStream_t stream);

int NCCLCommunicator::size() const {
  return size_;
}

int NCCLCommunicator::rank() const {
  return rank_;
}


/* CAPI **********************************************************************/

DGL_REGISTER_GLOBAL("cuda.nccl._CAPI_DGLNCCLGetUniqueId")
.set_body([] (DGLArgs args, DGLRetValue* rv) {
  *rv = NCCLUniqueIdRef(std::make_shared<NCCLUniqueId>());
});

DGL_REGISTER_GLOBAL("cuda.nccl._CAPI_DGLNCCLUniqueIdToString")
.set_body([] (DGLArgs args, DGLRetValue* rv) {
  NCCLUniqueIdRef idObj = args[0];
  *rv = idObj->ToString();
});

DGL_REGISTER_GLOBAL("cuda.nccl._CAPI_DGLNCCLUniqueIdFromString")
.set_body([] (DGLArgs args, DGLRetValue* rv) {
  const std::string str = args[0];

  NCCLUniqueIdRef ref(std::make_shared<NCCLUniqueId>());
  ref->FromString(str);
  *rv = ref;
});

DGL_REGISTER_GLOBAL("cuda.nccl._CAPI_DGLNCCLCreateComm")
.set_body([] (DGLArgs args, DGLRetValue* rv) {
  const int size = args[0];
  const int rank = args[1];
  NCCLUniqueIdRef idObj = args[2];

  *rv = NCCLCommunicatorRef(std::make_shared<NCCLCommunicator>(size, rank,
        idObj->Get()));
});

DGL_REGISTER_GLOBAL("cuda.nccl._CAPI_DGLNCCLSparseAllToAllPush")
.set_body([] (DGLArgs args, DGLRetValue* rv) {
  NCCLCommunicatorRef comm = args[0];
  IdArray in_idx = args[1];
  NDArray in_values = args[2];
  const int mode_id = args[3];

  List<ObjectRef> ret;
  ATEN_ID_TYPE_SWITCH(in_idx->dtype, IdType, {
    ATEN_DTYPE_SWITCH(in_values->dtype, DType, "values", {
      auto result = SparsePush<IdType, DType>(comm, in_idx, in_values, mode_id);
      ret.push_back(Value(MakeValue(result.first)));
      ret.push_back(Value(MakeValue(result.second)));
    });
  });

  *rv = ret;
});

DGL_REGISTER_GLOBAL("cuda.nccl._CAPI_DGLNCCLSparseAllToAllPull")
.set_body([] (DGLArgs args, DGLRetValue* rv) {
  NCCLCommunicatorRef comm = args[0];
  // the indexes this process is requesting from others
  IdArray req_idx = args[1];

  // the tensor this process has to fulfill other requests
  NDArray tensor = args[2];
  const int mode_id = args[3];

  ATEN_ID_TYPE_SWITCH(req_idx->dtype, IdType, {
    ATEN_DTYPE_SWITCH(tensor->dtype, DType, "values", {
      *rv = SparsePull<IdType, DType>(comm, req_idx, tensor, mode_id);
    });
  });
});


}  // namespace cuda
}  // namespace runtime
}  // namespace dgl



